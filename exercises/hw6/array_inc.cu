
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

template <typename T>
void alloc_bytes(T &ptr, size_t num_bytes){
  hipMallocManaged(&ptr, num_bytes);
}

__global__ void inc(int *array, size_t n){
  size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
  while (idx < n){
    array[idx]++;
    idx += blockDim.x*gridDim.x; // grid-stride loop
    }
}

const size_t  ds = 32ULL*1024ULL*1024ULL;

int main(){

  int *d_array;
  alloc_bytes(d_array, ds*sizeof(int));
  cudaCheckErrors("hipMalloc Error");

  memset(d_array, 0, ds*sizeof(int));
  cudaCheckErrors("hipMemcpy H->D Error");

  inc<<<256, 256>>>(d_array, ds);
  cudaCheckErrors("kernel launch error");

  for (int i = 0; i < ds; i++) 
    if (d_array[i] != 1) {printf("mismatch at %d, was: %d, expected: %d\n", i, d_array[i], 1); return -1;}
  printf("success!\n"); 
  return 0;
}
